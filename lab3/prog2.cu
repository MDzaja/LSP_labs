
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <math.h>

#define ARR_SIZE 1024 * 1024
#define COL_ROW_SIZE 1024

typedef struct SeqWrapper {
    int startingColumnIndex;
    int *arr;
} SeqWrapper;

__host__ __device__ int getElementByIndex(SeqWrapper *wrapper, int i) {
    int rel_column_i = i / COL_ROW_SIZE + wrapper->startingColumnIndex;
    int i_in_column = i % COL_ROW_SIZE;

    int index = wrapper->startingColumnIndex + rel_column_i + i_in_column * COL_ROW_SIZE;
    if(index >= ARR_SIZE || index < 0) 
        printf("Index out of bounds: %d\n", index);

    return wrapper->arr[index];
}

__host__ __device__ void setElementByIndex(SeqWrapper *wrapper, int i, int value) {
    int rel_column_i = i / COL_ROW_SIZE - wrapper->startingColumnIndex;
    int i_in_column = i % COL_ROW_SIZE;

    int index = wrapper->startingColumnIndex + rel_column_i+ i_in_column * COL_ROW_SIZE;
    if(index >= ARR_SIZE || index < 0) 
        printf("Index out of bounds: %d\n", index);

    wrapper->arr[index] = value;
}

void readFile(char *filename, int *arr, int size);
void transpose(int* matrix, int* transposed);
__device__ void mergeSort(SeqWrapper wrapper, int left, int right);
__device__ void merge(SeqWrapper wrapper, int left, int mid, int right);

__global__ void mergeSort_cuda(int *array, int iter)
{
    // 2D grid and block
    /*int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = blockDim.x * gridDim.x * y + x;*/
    // 1D grid and block
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    int subseq_size = COL_ROW_SIZE * pow(2, iter);

    SeqWrapper wrapper;
    int columnPerThread = pow(2, iter);
    wrapper.startingColumnIndex = idx * columnPerThread;
    wrapper.arr = array;

    // Perform the merge sort operation.
    mergeSort(wrapper, 0, subseq_size - 1);

    if (idx == 0) {
        printf("bla bla\n");
        printf("First element after merge sort: %d\n", array[0]);
    }
}

__global__ void merge_cuda(int *array, int iter)
{
    // 1D grid and block
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // m is the point where the array is divided into two subarrays
    int subseq_size = COL_ROW_SIZE * pow(2, iter);
    int mid = (subseq_size - 1) / 2;

    SeqWrapper wrapper;
    int column_per_thread = pow(2, iter);
    wrapper.startingColumnIndex = idx * column_per_thread;
    wrapper.arr = array;

    // Merge the two subarrays
    merge(wrapper, 0, mid, subseq_size-1);
}

int main(int argc, char **argv)
{
    // Check if the correct number of command line arguments are provided
    if (argc != 2)
    {
        printf("Usage: %s file1\n", argv[0]);
    }

    int *array, *transposed;
    // Allocate memory for the array.
    hipMallocManaged(&array, ARR_SIZE * sizeof(int));
    hipMallocManaged(&transposed, ARR_SIZE * sizeof(int));
    // Read the array from a file here.
    readFile(argv[1], array, ARR_SIZE);
    printf("First element: %d\n", array[0]);

    // Perform the sort.
    for (int iter = 0; iter <= 10; iter++)
    {
        int threads = 1024 / pow(2, iter);
        printf("Iteration %d, Threads: %d\n", iter+1, threads);
        if (iter == 0)
        {
            mergeSort_cuda<<<1, threads>>>(array, iter);
        }
        else
        {
            merge_cuda<<<1, threads>>>(array, iter);
        }
        //printf("First element after iteration %d: %d\n", iter+1, array[0]);
        // Wait for GPU to finish before accessing on host.
        hipDeviceSynchronize();
    }

    // Transpose the array
    /*printf("Transposing\n");
    transpose(array, transposed);
    printf("Transposed\n");*/
    printf("bzvz\n");
    printf("%d prvi clan\n", array[0]);
    SeqWrapper wrapper;
    wrapper.startingColumnIndex = 0;
    wrapper.arr = array;

    // Check if the sequence is properly sorted
    int i;
    for (i = 0; i < ARR_SIZE - 1; i++)
    {
        printf("prvi if\n");
        if (getElementByIndex(&wrapper, i) > getElementByIndex(&wrapper, i + 1))
        {
            printf("unutra\n");
            printf("Error in position %d between element %d and %d\n", i, getElementByIndex(&wrapper, i), getElementByIndex(&wrapper, i + 1));
            break;
        }
        printf("drugi\n");
    }
    if (i == (ARR_SIZE - 1))
    {
        printf("Everything is OK!\n");
    }

    // Free memory.
    hipFree(array);
    hipFree(transposed);

    return 0;
}

void readFile(char *filename, int *arr, int size)
{
    FILE *fp;
    int n = 0;

    // Open the binary file for reading
    fp = fopen(filename, "rb");
    if (fp == NULL)
    {
        printf("Error opening file\n");
        exit(1);
    }

    // Read the size of the array from the file
    int *true_size = (int *)malloc(sizeof(int));
    size_t bytes_read = fread(true_size, sizeof(int), 1, fp);
    if (size != *true_size)
    {
        printf("Error in size of the array. Array should have %d elements, but has %d elements.\n", size, *true_size);
        exit(1);
    }

    // Read the numbers from the file and store them in the array
    while (fread(&arr[n], sizeof(int), 1, fp) == 1)
    {
        n++;
    }

    // Close the file
    fclose(fp);
}

void transpose(int* matrix, int* transposed) {
    for (int i = 0; i < COL_ROW_SIZE; i++) {
        for (int j = 0; j < COL_ROW_SIZE; j++) {
            transposed[j*COL_ROW_SIZE + i] = matrix[i*COL_ROW_SIZE + j];
        }
    }
}

// Merge two sorted subarrays of the array
__device__ void merge(SeqWrapper wrapper, int left, int mid, int right)
{
    int n_left = mid - left + 1;
    int n_right = right - mid;

    int *left_arr = (int *)malloc(n_left * sizeof(int));
    int *right_arr = (int *)malloc(n_right * sizeof(int));

    for (int i = 0; i < n_left; i++)
        left_arr[i] = getElementByIndex(&wrapper, left + i);
    for (int j = 0; j < n_right; j++)
        right_arr[j] = getElementByIndex(&wrapper, mid + 1 + j);

    // Maintain current index of sub-arrays and main array
    int i, j, k;
    i = 0;
    j = 0;
    k = left;

    // Until we reach either end of either L or M, pick larger among
    // elements L and M and place them in the correct position at A[p..r]
    while (i < n_left && j < n_right)
    {
        if (left_arr[i] <= right_arr[j])
        {
            setElementByIndex(&wrapper, k++, left_arr[i++]);
        }
        else
        {
            setElementByIndex(&wrapper, k++, right_arr[j++]);
        }
    }

    // When we run out of elements in either L or M,
    // pick up the remaining elements and put in A[p..r]
    while (i < n_left)
    {
        setElementByIndex(&wrapper, k++, left_arr[i++]);
    }

    while (j < n_right)
    {
        setElementByIndex(&wrapper, k++, right_arr[j++]);
    }

    free(left_arr);
    free(right_arr);
}

// Divide the array into two subarrays, sort them and merge them
__device__ void mergeSort(SeqWrapper wrapper, int left, int right)
{
    if (left < right)
    {

        // m is the point where the array is divided into two subarrays
        int mid = left + (right - left) / 2;

        mergeSort(wrapper, left, mid);
        mergeSort(wrapper, mid + 1, right);

        // Merge the sorted subarrays
        merge(wrapper, left, mid, right);
    }
}

/*int calcIndex(int i, int idx, int iter) {
    int column_per_thread = pow(2, iter);
    int start_col_i = idx * column_per_thread;

    int rel_column_i = i / COL_ROW_SIZE - start_col_i;
    int i_in_column = i % COL_ROW_SIZE;

    return start_col_i + rel_column_i + i_in_column * COL_ROW_SIZE ;
}*/